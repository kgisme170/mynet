﻿#include "hip/hip_runtime.h"  
#include "hipblas.h"  

#include <time.h>  
#include <iostream>  

using namespace std;

int const M = 5;
int const N = 10;

int main()
{
	hipblasStatus_t status;
	float *h_A = (float*)malloc(N*M * sizeof(float));
	float *h_B = (float*)malloc(N*M * sizeof(float));
	float *h_C = (float*)malloc(M*M * sizeof(float));
	for (int i = 0; i < N*M; i++) {
		h_A[i] = (float)(rand() % 10 + 1);
		h_B[i] = (float)(rand() % 10 + 1);
	}
	for (int i = 0; i < N*M; i++) {
		cout << h_A[i] << " ";
		if ((i + 1) % N == 0) cout << endl;
	}
	cout << endl;
	for (int i = 0; i < N*M; i++) {
		cout << h_B[i] << " ";
		if ((i + 1) % M == 0) cout << endl;
	}
	cout << endl;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "CUBLAS 对象实例化出错" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}
	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, N*M * sizeof(float));
	hipMalloc((void**)&d_B, N*M * sizeof(float));
	hipMalloc((void**)&d_C, M*M * sizeof(float));
	hipblasSetVector(N*M, sizeof(float),	h_A, 1, d_A, 1);
	hipblasSetVector(N*M, sizeof(float),	h_B, 1,	d_B, 1);
	float a = 1; float b = 0;
	hipblasSgemm(
		handle,    // blas 库对象   
		HIPBLAS_OP_T,    // 矩阵 A 属性参数  
		HIPBLAS_OP_T,    // 矩阵 B 属性参数  
		M,    // A, C 的行数   
		M,    // B, C 的列数  
		N,    // A 的列数和 B 的行数  
		&a,    // 运算式的 α 值  
		d_A,    // A 在显存中的地址  
		N,    // lda  
		d_B,    // B 在显存中的地址  
		M,    // ldb  
		&b,    // 运算式的 β 值  
		d_C,    // C 在显存中的地址(结果矩阵)  
		M    // ldc  
	);
	hipblasGetVector(M*M, sizeof(float), d_C, 1,	h_C, 1);
	cout << "(A*B)T: " << endl;
	for (int i = 0; i < M*M; i++) {
		cout << h_C[i] << " ";
		if ((i + 1) % M == 0) cout << endl;
	}
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	hipblasDestroy(handle);
	return 0;
}
